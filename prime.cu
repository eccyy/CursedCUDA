
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string>
#include <vector>
#include <cstdint>



int main(int argc,char *argv[]){

	float x = std::stof(argv[1]);
	float n = sqrt(x); 
	
	bool * A = new bool[x];

	for (int i = 0; i < x; i++) {
        A[i] = 1;
    }

	for(int i = 2; i <= n; i++)
	{	
	
		if(A[i])
		{
		
			A[i * i] = false;
			for(int m = 1; ((m * i + i * i)) < x; m++)
			{	
				int j =(m * i + i * i);
				A[j] = false;
				
			}
		}
	}
	
	
	int primecount = 0;
	for(int i = 2; i <= x; i++)
	{
		if(A[i]==1)
		{
			//std::cout << i << " ";
			primecount++;
		}
		//std::cout << A[i] << " ";
	}
	
	std::cout << "primes: " << primecount;
	delete[] A;
	return 0;

}