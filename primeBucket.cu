#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <cmath>
#include <string>
#include <math.h>
#include <cstdint>

__global__ void primemod(bool *range, int n, int index, unsigned long int *primes)
{
    // what node/prime number will be running
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // run the ones allocated with prime numbers
    if (primes[i] != 0)
    {
        // j is the offset to the next multiple in the bucket
        for (int j = (primes[i] - ((n * index) % primes[i])); j <= n; j += primes[i])
        {

            range[j - 1] = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    unsigned long *primes;
    unsigned long x;
    int n;
    bool *range;
    unsigned long primeCount = 0;

    x = std::stof(argv[1]);
    n = sqrt(x);

    // Cuda blocks/threads
    int blockSize = 32;
    int numBlocks = ((n + blockSize - 1) / blockSize);

    // bucket part of the bucket sieve
    range = new bool[n];

    // use prime number theorem to estimate how many we need to store, intit to 0 so we know which ones haven't been computed
    int totalPrimes = (x / log(x));
    primes = new unsigned long[totalPrimes];

    hipMallocManaged(&range, sizeof(range) * n);
    hipMallocManaged(&primes, sizeof(primes) * (x / (log(x))));
    std::memset(primes, 0, x / (log(x)));

    // find first bucket of primes
    bool *A = new bool[n];
    memset(A, 1, n);
    for (int i = 2; i < n; i++)
    {

        if (A[i])
        {
            primes[primeCount] = i;
            primeCount++;

            if ((i * i) < n)
            {
                A[i * i] = false;
            }
            for (int m = 1; (m * i + i * i) < n; m++)
            {
                int j = (m * i + i * i);
                A[j] = false;
            }
        }
    }

    // sieve the primes inside the bucket then count them
    for (int index = 1; (index * n) <= x + n; index++)
    {

        std::memset(range, 1, n);

        // run a kernel to siev the primes in the current bucket 
        primemod<<<blockSize, numBlocks>>>(range, n, index, primes);
        hipDeviceSynchronize();

        // update the new primes
        for (int i = 0; i < n; i++)
        {
            // check the bucket for primes
            if (range[i] && ((index * n) + i + 1) <= x)
            {
                // count and add the primes in the shared array
                primeCount++;
                primes[primeCount - 1] = ((index * n) + i + 1);
            }
        }

    }

    std::cout << "primes: " << primeCount;

    hipFree(primes);
    hipFree(range);
    delete[] A;
    return 0;
}
