
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string>
#include <math.h>
#include <cstdint>

__global__
void primemod(bool* a, float x, float n)
{	
	int i = threadIdx.x + blockIdx.x * blockDim.x + 2;	  

	if(i<x && a[i])
	{
		for(int m = 2; (m * i) <= x; m++)
		{
			if(!a[i])
			{

				break;
			}
			a[i * m] = false;
		}
		
	}
	return;
}




int main(int argc,char *argv[]){

	float x = std::stof(argv[1]);
	float n = sqrt(x); 
	
	bool * A = new bool[x];
	//bool A[x];
	hipMallocManaged(&A,2*x*sizeof(bool));

	for (int i = 0; i < x; i++) {
        A[i] = 1;
    }
	
	std::cout<<"sqrt of " << x << " is " << n << "\n";

	
	int blockSize = 8;
	int numBlocks = (n + blockSize - 1) / blockSize;
	
	std::cout <<" numblocks " << numBlocks << " ";
	
	primemod<<<numBlocks,blockSize>>>(A,x,n);


	hipDeviceSynchronize();
	
	int primecount = 0;
	for(int i = 2; i <= x; i++)
	{
		if(A[i]==1)
		{
			//std::cout << i << " ";
			primecount++;
		}
		//std::cout << A[i] << " ";
	}
	
	std::cout << "primes: " << primecount;
	
	hipFree(A);
	delete[] A;
	return 0;

}