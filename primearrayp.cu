
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string>
#include <math.h>
#include <cstdint>

__global__
void primemod(bool* a, float x)
{	
	
	int i = threadIdx.x + blockIdx.x * blockDim.x + 2;
	if(i<x)
	{
		for(int k = i; k < x; k++)
			{
				if(k%i==0 && i!=k)
				{
					a[k]=false;
				}
			}
	}
	return;
}




int main(int argc,char *argv[]){

	float x = std::stof(argv[1]);
	float n = sqrt(x); 
	
	bool * A = new bool[x];
	//bool A[x];
	hipMallocManaged(&A,2*x*sizeof(bool));

	for (int i = 0; i < x; i++) {
        A[i] = 1;
    }
	
	//std::cout<<"sqrt of " << x << " is " << n << "\n";

	
	int blockSize = 1024;
	int numBlocks = (n + blockSize - 1) / blockSize;
	
	primemod<<<numBlocks,blockSize>>>(A,x);

	hipDeviceSynchronize();
	
	int primecount = 0;
	for(int i = 2; i < x; i++)
	{
		if(A[i]==1)
		{
			//std::cout << i << " ";
			primecount++;
		}
		//std::cout << A[i] << " ";
	}
	
	std::cout << "primes: " << primecount;
	
	hipFree(A);
	delete[] A;
	return 0;

}